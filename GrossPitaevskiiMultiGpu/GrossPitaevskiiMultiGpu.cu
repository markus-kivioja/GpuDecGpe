#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

#include "VortexState.hpp"
#include "Output/Picture.hpp"
#include "Output/Text.hpp"
#include "Types/Complex.hpp"
#include "Types/Random.hpp"
#include "Mesh/DelaunayMesh.hpp"

#include <iostream>
#include <sstream>
#include <chrono>

#include <mesh.h>

ddouble RATIO = 0.1;
ddouble KAPPA = 20;
ddouble G = 5000;

#define LOAD_STATE_FROM_DISK 1
#define SAVE_PICTURE 0
#define SAVE_VOLUME 0

#define THREAD_BLOCK_X 16
#define THREAD_BLOCK_Y 1
#define THREAD_BLOCK_Z 1

#define WARP_SIZE 32

ddouble potentialRZ(const ddouble r, const ddouble z)
{
	return 0.5 * (r * r + RATIO * RATIO * z * z);
}

ddouble potentialV3(const Vector3& p)
{
	return 0.5 * (p.x * p.x + p.y * p.y + RATIO * RATIO * p.z * p.z);
}

bool saveVolumeMap(const std::string& path, const Buffer<ushort>& vol, const uint xsize, const uint ysize, const uint zsize, const Vector3& h)
{
	Text rawpath;
	rawpath << path << ".raw";

	// save raw
	std::ofstream fs(rawpath.str().c_str(), std::ios_base::binary | std::ios::trunc);
	if (fs.fail()) return false;
	fs.write((char*)&vol[0], 2 * xsize * ysize * zsize);
	fs.close();

	// save header
	Text text;

	text << "ObjectType              = Image" << std::endl;
	text << "NDims                   = 3" << std::endl;
	text << "BinaryData              = True" << std::endl;
	text << "CompressedData          = False" << std::endl;
	text << "BinaryDataByteOrderMSB  = False" << std::endl;
	text << "TransformMatrix         = 1 0 0 0 1 0 0 0 1" << std::endl;
	text << "Offset                  = " << -0.5 * xsize * h.x << " " << -0.5 * ysize * h.y << " " << -0.5 * zsize * h.z << std::endl;
	text << "CenterOfRotation        = 0 0 0" << std::endl;
	text << "DimSize                 = " << xsize << " " << ysize << " " << zsize << std::endl;
	text << "ElementSpacing          = " << h.x << " " << h.y << " " << h.z << std::endl;
	text << "ElementNumberOfChannels = 1" << std::endl;
	text << "ElementType             = MET_USHORT" << std::endl;
	text << "ElementDataFile         = " << rawpath.str() << std::endl;
	text.save(path);
	return true;
}

struct BlockPsis
{
	double2 values[VALUES_IN_BLOCK];
};

struct LdsBlockPsis
{
	double3 values[VALUES_IN_BLOCK];
};

struct BlockPots
{
	double values[VALUES_IN_BLOCK];
};

struct PitchedPtr
{
	char* ptr;
	size_t pitch;
	size_t slicePitch;
};

// Arithmetic operators for cuda vector types
inline __host__ __device__ double2 operator+(double2 a, double2 b)
{
	return make_double2(a.x + b.x, a.y + b.y);
}
inline __host__ __device__ double2 operator-(double2 a, double2 b)
{
	return make_double2(a.x - b.x, a.y - b.y);
}
inline __host__ __device__ void operator+=(double2& a, double2 b)
{
	a.x += b.x;
	a.y += b.y;
}
inline __host__ __device__ double2 operator*(double b, double2 a)
{
	return make_double2(b * a.x, b * a.y);
}

__device__ const uint perms[VALUES_IN_BLOCK][FACE_COUNT] = {
		{3, 2, 0, 1}, // 0
		{2, 0, 1, 3}, // 1
		{1, 0, 2, 3}, // 2
		{1, 0, 2, 3}, // 3 
		{2, 0, 1, 3}, // 4
		{3, 1, 0, 2}, // 5
		{0, 1, 2, 3}, // 6
		{0, 1, 2, 3}, // 7
		{3, 1, 0, 2}, // 8
		{2, 3, 0, 1}, // 9
		{2, 3, 0, 1}, // 10
		{3, 2, 0, 1}  // 11
};

__global__ void update(PitchedPtr nextStep, PitchedPtr prevStep, PitchedPtr potentials, int3* blockDirs, int* valueInds, double* hodges, double g, uint3 dimensions)
{
	size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	size_t zid = blockIdx.z * blockDim.z + threadIdx.z;
	size_t dataXid = xid / VALUES_IN_BLOCK; // One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on z-axis)

	// Kill the leftover threads but leave threads for the additional zero buffer at the edges
	if (dataXid > dimensions.x || yid > dimensions.y || zid > dimensions.z)
	{
		return;
	}

	__shared__ LdsBlockPsis ldsPrevPsis[THREAD_BLOCK_Z * THREAD_BLOCK_Y * THREAD_BLOCK_X];
	size_t threadIdxInBlock = threadIdx.z * THREAD_BLOCK_Y * THREAD_BLOCK_X + threadIdx.y * THREAD_BLOCK_X + threadIdx.x / VALUES_IN_BLOCK;

	size_t dualNodeId = xid % VALUES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on z-axis)

	char* prevPsi = prevStep.ptr + prevStep.slicePitch * zid + prevStep.pitch * yid + sizeof(BlockPsis) * dataXid;
	BlockPots* pot = (BlockPots*)(potentials.ptr + potentials.slicePitch * zid + potentials.pitch * yid) + dataXid;
	BlockPsis* nextPsi = (BlockPsis*)(nextStep.ptr + nextStep.slicePitch * zid + nextStep.pitch * yid) + dataXid;
	double2 prev = ((BlockPsis*)prevPsi)->values[dualNodeId];
	ldsPrevPsis[threadIdxInBlock].values[dualNodeId] = make_double3(prev.x, prev.y, 0);
	double normsq = prev.x * prev.x + prev.y * prev.y;

	// Kill also the leftover edge threads
	if (dataXid == dimensions.x || yid == dimensions.y || zid == dimensions.z)
	{
		return;
	}

	uint idxInWarp = threadIdxInBlock % WARP_SIZE;

	uint primaryFaceStart = dualNodeId * FACE_COUNT;
	double2 sum = make_double2(0, 0);

	__syncthreads();
#pragma unroll
	for (int i = 0; i < FACE_COUNT; ++i)
	{
		uint primaryFace = primaryFaceStart + perms[dualNodeId][i];

		int neighbourX = threadIdx.x / VALUES_IN_BLOCK + blockDirs[primaryFace].x;
		int neighbourY = threadIdx.y + blockDirs[primaryFace].y;
		int neighbourZ = threadIdx.z + blockDirs[primaryFace].z;

		double2 neighbourPsi;

		// Read from the local shared memory
		if ((0 <= neighbourX) && (neighbourX < THREAD_BLOCK_X) &&
			(0 <= neighbourY) && (neighbourY < THREAD_BLOCK_Y) &&
			(0 <= neighbourZ) && (neighbourZ < THREAD_BLOCK_Z))
		{
			int neighbourIdx = neighbourZ * THREAD_BLOCK_Y * THREAD_BLOCK_X + neighbourY * THREAD_BLOCK_X + neighbourX;
			double3 temp = ldsPrevPsis[neighbourIdx].values[valueInds[primaryFace]];
			neighbourPsi = make_double2(temp.x, temp.y);
		}
		else // Read from the global memory
		{
			int offset = blockDirs[primaryFace].z * prevStep.slicePitch + blockDirs[primaryFace].y * prevStep.pitch + blockDirs[primaryFace].x * sizeof(BlockPsis);
			neighbourPsi = ((BlockPsis*)(prevPsi + offset))->values[valueInds[primaryFace]];
		}
		sum += hodges[primaryFace] * (neighbourPsi - prev);
	}

	sum += (pot->values[dualNodeId] + g * normsq) * prev;

	nextPsi->values[dualNodeId] += make_double2(sum.y, -sum.x);
};

ddouble integrateInTime(const VortexState& state, const ddouble block_scale, const Vector3& minp, const Vector3& maxp, const ddouble iteration_period, const uint number_of_iterations, uint gpuCount, bool enablePeerAccess = true)
{
	uint i, j, k, l;

	// find dimensions
	const Vector3 domain = maxp - minp;
	const uint xsize = uint(domain.x / (block_scale * BLOCK_WIDTH.x)) + 1;
	const uint ysize = uint(domain.y / (block_scale * BLOCK_WIDTH.y)) + 1;
	const uint zsize = uint(domain.z / (block_scale * BLOCK_WIDTH.z)) + 1;
	const Vector3 p0 = 0.5 * (minp + maxp - block_scale * Vector3(BLOCK_WIDTH.x * xsize, BLOCK_WIDTH.y * ysize, BLOCK_WIDTH.z * zsize));

	std::vector<uint> zSizes(gpuCount);
	uint zRemainder = zsize % gpuCount;
	for (uint gpuIdx = 0; gpuIdx < gpuCount; ++gpuIdx)
	{
		zSizes[gpuIdx] = zsize / gpuCount;
		if (zRemainder)
		{
			zSizes[gpuIdx]++;
			zRemainder--;
		}
		//std::cout << "GPU " << gpuIdx << " z-size: " << zSizes[gpuIdx] << ", ";
	}
	//std::cout << std::endl;

	std::cout << "Problem size: " << xsize << ", " << ysize << ", " << zsize << std::endl;
	//std::cout << "z-slice size: " << xsize << ", " << ysize << std::endl;

	// find relative circumcenters for each body element
	Buffer<Vector3> bpos;
	getPositions(bpos);

	// compute discrete dimensions
	const uint bsize = bpos.size(); // number of values inside a block
	const uint bxsize = (xsize + 1) * bsize; // number of values on x-row
	const uint bxysize = (ysize + 1) * bxsize; // number of values on xy-plane
	const uint ii0 = (IS_3D ? bxysize : 0) + bxsize + bsize; // reserved zeros in the beginning of value table
	const uint vsize = ii0 + (IS_3D ? zsize + 1 : zsize) * bxysize; // total number of values

	//std::cout << "bodies = " << xsize * ysize * zsize * bsize << std::endl;

	// initialize stationary state
	Buffer<Complex> Psi0(vsize, Complex(0, 0)); // initial discrete wave function
	Buffer<ddouble> pot(vsize, 0.0); // discrete potential multiplied by time step size
	ddouble g = state.getG(); // effective interaction strength
	ddouble maxpot = 0.0; // maximal value of potential
	for (k = 0; k < zsize; k++)
	{
		for (j = 0; j < ysize; j++)
		{
			for (i = 0; i < xsize; i++)
			{
				for (l = 0; l < bsize; l++)
				{
					const uint ii = ii0 + k * bxysize + j * bxsize + i * bsize + l;
					const Vector3 p(p0.x + block_scale * (i * BLOCK_WIDTH.x + bpos[l].x), p0.y + block_scale * (j * BLOCK_WIDTH.y + bpos[l].y), p0.z + block_scale * (k * BLOCK_WIDTH.z + bpos[l].z)); // position
					Psi0[ii] = state.getPsi(p);
					pot[ii] = potentialV3(p);
					const ddouble poti = pot[ii] + g * Psi0[ii].normsq();
					if (poti > maxpot) maxpot = poti;
				}
			}
		}
	}

	// find terms for laplacian
	Buffer<int3> blockDirs;
	Buffer<int> valueInds;
	Buffer<ddouble> hodges;
	ddouble lapfac = -0.5 * getLaplacian(blockDirs, valueInds, hodges) / (block_scale * block_scale);
	const uint lapsize = blockDirs.size() / bsize;
	ddouble lapfac0 = lapsize * (-lapfac);

	// compute time step size
	const uint steps_per_iteration = 691; // uint(iteration_period * (maxpot + lapfac0)) + 1; // number of time steps per iteration period
	const ddouble time_step_size = iteration_period / ddouble(steps_per_iteration); // time step in time units

	std::cout << "steps_per_iteration = " << steps_per_iteration << std::endl;

	// multiply terms with time_step_size
	g *= time_step_size;
	lapfac *= time_step_size;
	lapfac0 *= time_step_size;
	for (i = 0; i < vsize; i++) pot[i] *= time_step_size;
	for (int i = 0; i < hodges.size(); ++i) hodges[i] = -0.5 * hodges[i] / (block_scale * block_scale) * time_step_size;

	// Initialize host memory
	size_t dxsize = xsize + 2; // One element buffer to both ends
	size_t dysize = ysize + 2;
	size_t hostSize = dxsize * dysize * (zsize + 2);
	BlockPsis* h_evenPsi;// = new BlockPsis[dxsize * dysize * (zsize + 2)];
	BlockPsis* h_oddPsi;// = new BlockPsis[dxsize * dysize * (zsize + 2)];
	BlockPots* h_pot;// = new BlockPots[dxsize * dysize * (zsize + 2)];
	checkCudaErrors(hipHostMalloc(&h_evenPsi, hostSize * sizeof(BlockPsis)));
	checkCudaErrors(hipHostMalloc(&h_oddPsi, hostSize * sizeof(BlockPsis)));
	checkCudaErrors(hipHostMalloc(&h_pot, hostSize * sizeof(BlockPots)));
	memset(h_evenPsi, 0, hostSize * sizeof(BlockPsis));
	memset(h_oddPsi, 0, hostSize * sizeof(BlockPsis));
	memset(h_pot, 0, hostSize * sizeof(BlockPots));

	// initialize discrete field
	const Complex oddPhase = state.getPhase(-0.5 * time_step_size);
	Random rnd(54363);
	for (k = 0; k < zsize; k++)
	{
		for (j = 0; j < ysize; j++)
		{
			for (i = 0; i < xsize; i++)
			{
				for (l = 0; l < bsize; l++)
				{
					const uint srcI = ii0 + k * bxysize + j * bxsize + i * bsize + l;
					const uint dstI = (k + 1) * dxsize * dysize + (j + 1) * dxsize + (i + 1);
					const Vector2 c = 0.01 * rnd.getUniformCircle();
					const Complex noise(c.x + 1.0, c.y);
					const Complex noisedPsi = Psi0[srcI] * noise;
					double2 even = make_double2(noisedPsi.r, noisedPsi.i);
					h_evenPsi[dstI].values[l] = even;
					h_oddPsi[dstI].values[l] = make_double2(oddPhase.r * even.x - oddPhase.i * even.y,
						oddPhase.i * even.x + oddPhase.r * even.y);
					h_pot[dstI].values[l] = pot[srcI];
				}
			}
		}
	}

	// Initialize device memory
	std::vector<hipPitchedPtr> d_cudaEvenPsis(gpuCount);
	std::vector<hipPitchedPtr> d_cudaOddPsis(gpuCount);
	std::vector<hipPitchedPtr> d_cudaPots(gpuCount);
	std::vector<PitchedPtr> d_evenPsis(gpuCount);
	std::vector<PitchedPtr> d_oddPsis(gpuCount);
	std::vector<PitchedPtr> d_pots(gpuCount);
	std::vector<int3*> d_blockDirs(gpuCount);
	std::vector<int*> d_valueInds(gpuCount);
	std::vector<ddouble*> d_hodges(gpuCount);
	std::vector<hipPitchedPtr> h_cudaEvenPsis(gpuCount);
	std::vector<hipPitchedPtr> h_cudaOddPsis(gpuCount);
	std::vector<hipPitchedPtr> h_cudaPots(gpuCount);
	std::vector<hipExtent> psiExtents(gpuCount);

	std::vector<size_t> dzSizes(gpuCount);
	for (uint gpuIdx = 0; gpuIdx < gpuCount; ++gpuIdx)
	{
		dzSizes[gpuIdx] = zSizes[gpuIdx] + 2;

		psiExtents[gpuIdx] = make_hipExtent(dxsize * sizeof(BlockPsis), dysize, dzSizes[gpuIdx]);
		hipExtent potExtent = make_hipExtent(dxsize * sizeof(BlockPots), dysize, dzSizes[gpuIdx]);

		hipSetDevice(gpuIdx);

		if (enablePeerAccess)
		{
			for (uint peerGpu = 0; peerGpu < gpuCount; ++peerGpu)
			{
				if (peerGpu != gpuIdx)
				{
					int canAccessPeer;
					hipDeviceCanAccessPeer(&canAccessPeer, gpuIdx, peerGpu);
					if ((canAccessPeer == 1) && hipDeviceEnablePeerAccess(peerGpu, 0) == hipSuccess)
					{
						//std::cout << "GPU " << gpuIdx << " can access GPU " << peerGpu << std::endl;
					}
					else
					{
						//std::cout << "GPU " << gpuIdx << " can NOT access GPU " << peerGpu << std::endl;
					}
				}
			}
		}

		checkCudaErrors(hipMalloc3D(&d_cudaEvenPsis[gpuIdx], psiExtents[gpuIdx]));
		checkCudaErrors(hipMalloc3D(&d_cudaOddPsis[gpuIdx], psiExtents[gpuIdx]));
		checkCudaErrors(hipMalloc3D(&d_cudaPots[gpuIdx], potExtent));

		// Offsets are for the zero valued padding on the edges, offset = z + y + x in bytes
		size_t offset = d_cudaEvenPsis[gpuIdx].pitch * dysize + d_cudaEvenPsis[gpuIdx].pitch + sizeof(BlockPsis);
		size_t potOffset = d_cudaPots[gpuIdx].pitch * dysize + d_cudaPots[gpuIdx].pitch + sizeof(BlockPots);
		PitchedPtr d_evenPsi = { (char*)d_cudaEvenPsis[gpuIdx].ptr + offset, d_cudaEvenPsis[gpuIdx].pitch, d_cudaEvenPsis[gpuIdx].pitch * dysize };
		PitchedPtr d_oddPsi = { (char*)d_cudaOddPsis[gpuIdx].ptr + offset, d_cudaOddPsis[gpuIdx].pitch, d_cudaOddPsis[gpuIdx].pitch * dysize };
		PitchedPtr d_pot = { (char*)d_cudaPots[gpuIdx].ptr + potOffset, d_cudaPots[gpuIdx].pitch, d_cudaPots[gpuIdx].pitch * dysize };
		d_evenPsis[gpuIdx] = d_evenPsi;
		d_oddPsis[gpuIdx] = d_oddPsi;
		d_pots[gpuIdx] = d_pot;

		checkCudaErrors(hipMalloc(&d_blockDirs[gpuIdx], blockDirs.size() * sizeof(int3)));
		checkCudaErrors(hipMalloc(&d_valueInds[gpuIdx], valueInds.size() * sizeof(int)));
		checkCudaErrors(hipMalloc(&d_hodges[gpuIdx], hodges.size() * sizeof(ddouble)));

		bool first = (gpuIdx == 0);

		h_cudaEvenPsis[gpuIdx].ptr = first ? h_evenPsi : ((BlockPsis*)h_cudaEvenPsis[gpuIdx - 1].ptr) + dxsize * dysize * (dzSizes[gpuIdx - 1] - 2);
		h_cudaEvenPsis[gpuIdx].pitch = dxsize * sizeof(BlockPsis);
		h_cudaEvenPsis[gpuIdx].xsize = d_cudaEvenPsis[gpuIdx].xsize;
		h_cudaEvenPsis[gpuIdx].ysize = d_cudaEvenPsis[gpuIdx].ysize;

		h_cudaOddPsis[gpuIdx].ptr = first ? h_oddPsi : ((BlockPsis*)h_cudaOddPsis[gpuIdx - 1].ptr) + dxsize * dysize * (dzSizes[gpuIdx - 1] - 2);
		h_cudaOddPsis[gpuIdx].pitch = dxsize * sizeof(BlockPsis);
		h_cudaOddPsis[gpuIdx].xsize = d_cudaOddPsis[gpuIdx].xsize;
		h_cudaOddPsis[gpuIdx].ysize = d_cudaOddPsis[gpuIdx].ysize;

		h_cudaPots[gpuIdx].ptr = first ? h_pot : ((BlockPots*)h_cudaPots[gpuIdx - 1].ptr) + dxsize * dysize * (dzSizes[gpuIdx - 1] - 2);
		h_cudaPots[gpuIdx].pitch = dxsize * sizeof(BlockPots);
		h_cudaPots[gpuIdx].xsize = d_cudaPots[gpuIdx].xsize;
		h_cudaPots[gpuIdx].ysize = d_cudaPots[gpuIdx].ysize;

		// Copy from host memory to device memory
		hipMemcpy3DParms evenPsiParams = { 0 };
		hipMemcpy3DParms oddPsiParams = { 0 };
		hipMemcpy3DParms potParams = { 0 };

		evenPsiParams.srcPtr = h_cudaEvenPsis[gpuIdx];
		evenPsiParams.dstPtr = d_cudaEvenPsis[gpuIdx];
		evenPsiParams.extent = psiExtents[gpuIdx];
		evenPsiParams.kind = hipMemcpyHostToDevice;

		oddPsiParams.srcPtr = h_cudaOddPsis[gpuIdx];
		oddPsiParams.dstPtr = d_cudaOddPsis[gpuIdx];
		oddPsiParams.extent = psiExtents[gpuIdx];
		oddPsiParams.kind = hipMemcpyHostToDevice;

		potParams.srcPtr = h_cudaPots[gpuIdx];
		potParams.dstPtr = d_cudaPots[gpuIdx];
		potParams.extent = potExtent;
		potParams.kind = hipMemcpyHostToDevice;

		checkCudaErrors(hipMemcpy3DAsync(&evenPsiParams));
		checkCudaErrors(hipMemcpy3DAsync(&oddPsiParams));
		checkCudaErrors(hipMemcpy3DAsync(&potParams));
		checkCudaErrors(hipMemcpy(d_blockDirs[gpuIdx], &blockDirs[0], blockDirs.size() * sizeof(int3), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_valueInds[gpuIdx], &valueInds[0], valueInds.size() * sizeof(int), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_hodges[gpuIdx], &hodges[0], hodges.size() * sizeof(ddouble), hipMemcpyHostToDevice));

		hipDeviceSynchronize();
	}

	// Clear host memory after data has been copied to devices
	Psi0.clear();
	pot.clear();
	bpos.clear();
	blockDirs.clear();
	valueInds.clear();
	hodges.clear();
	hipHostFree(h_oddPsi);
	hipHostFree(h_pot);
#if !(SAVE_PICTURE || SAVE_VOLUME)
	hipHostFree(h_evenPsi);
#endif

	// Integrate in time
	dim3 dimBlock(THREAD_BLOCK_X * VALUES_IN_BLOCK, THREAD_BLOCK_Y, THREAD_BLOCK_Z);
	std::vector<dim3> dimGrids(gpuCount);
	std::vector<uint3> dimensions(gpuCount);
	for (uint gpuIdx = 0; gpuIdx < gpuCount; ++gpuIdx)
	{
		dimensions[gpuIdx] = make_uint3(xsize, ysize, zSizes[gpuIdx]);
		dimGrids[gpuIdx] = dim3((xsize + THREAD_BLOCK_X - 1) / THREAD_BLOCK_X,
			(ysize + THREAD_BLOCK_Y - 1) / THREAD_BLOCK_Y,
			(zSizes[gpuIdx] + THREAD_BLOCK_Z - 1) / THREAD_BLOCK_Z);
	}

	std::vector<hipMemcpy3DParms> evenMemcpiesFrom(gpuCount - 1, { 0 });
	std::vector<hipMemcpy3DParms> evenMemcpiesTo(gpuCount - 1, { 0 });
	std::vector<hipMemcpy3DParms> oddMemcpiesFrom(gpuCount - 1, { 0 });
	std::vector<hipMemcpy3DParms> oddMemcpiesTo(gpuCount - 1, { 0 });
	
	hipExtent oneSliceExtent = make_hipExtent(dxsize * sizeof(BlockPsis), dysize, 1);
	for (uint gpuIdx = 0; gpuIdx < gpuCount - 1; ++gpuIdx)
	{
		hipPitchedPtr evenFromSrc = d_cudaEvenPsis[gpuIdx];
		hipPitchedPtr evenFromDst = d_cudaEvenPsis[gpuIdx + 1];
		hipPitchedPtr evenToSrc = d_cudaEvenPsis[gpuIdx + 1];
		hipPitchedPtr evenToDst = d_cudaEvenPsis[gpuIdx];
	
		hipPitchedPtr oddFromSrc = d_cudaOddPsis[gpuIdx];
		hipPitchedPtr oddFromDst = d_cudaOddPsis[gpuIdx + 1];
		hipPitchedPtr oddToSrc = d_cudaOddPsis[gpuIdx + 1];
		hipPitchedPtr oddToDst = d_cudaOddPsis[gpuIdx];
	
		evenFromSrc.ptr = ((char*)evenFromSrc.ptr) + d_evenPsis[gpuIdx].slicePitch * (dzSizes[gpuIdx] - 2);
		evenToDst.ptr = ((char*)evenToDst.ptr) + d_evenPsis[gpuIdx].slicePitch * (dzSizes[gpuIdx] - 1);
		evenToSrc.ptr = ((char*)evenToSrc.ptr) + d_evenPsis[gpuIdx + 1].slicePitch * 1;
		evenFromDst.ptr = ((char*)evenFromDst.ptr) + d_evenPsis[gpuIdx + 1].slicePitch * 0;
	
		oddFromSrc.ptr = ((char*)oddFromSrc.ptr) + d_oddPsis[gpuIdx].slicePitch * (dzSizes[gpuIdx] - 2);
		oddToDst.ptr = ((char*)oddToDst.ptr) + d_oddPsis[gpuIdx].slicePitch * (dzSizes[gpuIdx] - 1);
		oddToSrc.ptr = ((char*)oddToSrc.ptr) + d_oddPsis[gpuIdx + 1].slicePitch * 1;
		oddFromDst.ptr = ((char*)oddFromDst.ptr) + d_oddPsis[gpuIdx + 1].slicePitch * 0;

		evenMemcpiesFrom[gpuIdx].srcPtr = evenFromSrc;
		evenMemcpiesFrom[gpuIdx].dstPtr = evenFromDst;
		evenMemcpiesFrom[gpuIdx].extent = oneSliceExtent;
		evenMemcpiesFrom[gpuIdx].kind = hipMemcpyDefault;

		evenMemcpiesTo[gpuIdx].srcPtr = evenToSrc;
		evenMemcpiesTo[gpuIdx].dstPtr = evenToDst;
		evenMemcpiesTo[gpuIdx].extent = oneSliceExtent;
		evenMemcpiesTo[gpuIdx].kind = hipMemcpyDefault;

		oddMemcpiesFrom[gpuIdx].srcPtr = oddFromSrc;
		oddMemcpiesFrom[gpuIdx].dstPtr = oddFromDst;
		oddMemcpiesFrom[gpuIdx].extent = oneSliceExtent;
		oddMemcpiesFrom[gpuIdx].kind = hipMemcpyDefault;

		oddMemcpiesTo[gpuIdx].srcPtr = oddToSrc;
		oddMemcpiesTo[gpuIdx].dstPtr = oddToDst;
		oddMemcpiesTo[gpuIdx].extent = oneSliceExtent;
		oddMemcpiesTo[gpuIdx].kind = hipMemcpyDefault;
	}

	struct StreamsAndEvents
	{
		hipStream_t backwardsStream;
		hipStream_t forwardsStream;
		hipStream_t kernelStream;

		hipEvent_t backwardsEvent;
		hipEvent_t forwardsEvent;
		hipEvent_t kernelEvent;
	};
	std::vector<StreamsAndEvents> streamAndEvents(gpuCount);
	for (uint gpuIdx = 0; gpuIdx < gpuCount; ++gpuIdx)
	{
		hipSetDevice(gpuIdx);
		hipStreamCreate(&streamAndEvents[gpuIdx].backwardsStream);
		hipStreamCreate(&streamAndEvents[gpuIdx].forwardsStream);
		hipStreamCreate(&streamAndEvents[gpuIdx].kernelStream);

		hipEventCreate(&streamAndEvents[gpuIdx].backwardsEvent);
		hipEventCreate(&streamAndEvents[gpuIdx].forwardsEvent);
		hipEventCreate(&streamAndEvents[gpuIdx].kernelEvent);

		if (gpuIdx < gpuCount - 1)
		{
			hipEventRecord(streamAndEvents[gpuIdx].forwardsEvent, streamAndEvents[gpuIdx].forwardsStream);
		}
		if (0 < gpuIdx)
		{
			hipEventRecord(streamAndEvents[gpuIdx].backwardsEvent, streamAndEvents[gpuIdx].backwardsStream);
		}
	}

#if SAVE_PICTURE || SAVE_VOLUME
	std::vector<hipMemcpy3DParms> evenPsiBackParams(gpuCount, { 0 });
	for (uint gpuIdx = 0; gpuIdx < gpuCount; ++gpuIdx)
	{
		d_cudaEvenPsis[gpuIdx].ptr = ((char*)d_cudaEvenPsis[gpuIdx].ptr) + d_evenPsis[gpuIdx].slicePitch;
		h_cudaEvenPsis[gpuIdx].ptr = ((BlockPsis*)h_cudaEvenPsis[gpuIdx].ptr) + dxsize * dysize;
		psiExtents[gpuIdx].depth -= 2;

		evenPsiBackParams[gpuIdx].srcPtr = d_cudaEvenPsis[gpuIdx];
		evenPsiBackParams[gpuIdx].dstPtr = h_cudaEvenPsis[gpuIdx];
		evenPsiBackParams[gpuIdx].extent = psiExtents[gpuIdx];
		evenPsiBackParams[gpuIdx].kind = hipMemcpyDeviceToHost;
	}
#endif
	double2 lapfacs = make_double2(lapfac, lapfac0);

	uint iter = 0;
	auto startTime = std::chrono::system_clock::now();
	while (true)
	{
#if SAVE_PICTURE || SAVE_VOLUME
		//hipDeviceSynchronize();
#endif

#if SAVE_PICTURE
		// draw picture
		Picture pic(dxsize, dysize);
		k = zsize / 2 + 1;
		for (j = 0; j < dysize; j++)
		{
			for (i = 0; i < dxsize; i++)
			{
				const uint idx = k * dxsize * dysize + j * dxsize + i;
				double norm = sqrt(h_evenPsi[idx].values[0].x * h_evenPsi[idx].values[0].x + h_evenPsi[idx].values[0].y * h_evenPsi[idx].values[0].y);

				pic.setColor(i, j, 5.0 * Vector4(h_evenPsi[idx].values[0].x, norm, h_evenPsi[idx].values[0].y, 1.0));
			}
		}
		std::ostringstream picpath;
		picpath << "tulokset/kuva" << iter << ".bmp";
		pic.save(picpath.str(), false);
#endif

#if SAVE_VOLUME
		// save volume map
		const ddouble fmax = state.searchFunctionMax();
		const ddouble unit = 60000.0 / (bsize * fmax * fmax);
		Buffer<ushort> vol(dxsize * dysize * (zsize + 2));
		for (k = 0; k < (zsize + 2); k++)
		{
			for (j = 0; j < dysize; j++)
			{
				for (i = 0; i < dxsize; i++)
				{
					const uint idx = k * dxsize * dysize + j * dxsize + i;
					ddouble sum = 0.0;
					for (l = 0; l < bsize; l++)
					{
						sum += h_evenPsi[idx].values[0].x * h_evenPsi[idx].values[0].x + h_evenPsi[idx].values[0].y * h_evenPsi[idx].values[0].y;
					}
					sum *= unit;
					vol[idx] = (sum > 65535.0 ? 65535 : ushort(sum));
				}
			}
		}
		Text volpath;
		volpath << "volume" << iter << ".mhd";
		saveVolumeMap(volpath.str(), vol, dxsize, dysize, (zsize + 2), block_scale * BLOCK_WIDTH);
#endif

		// finish iteration
		if (++iter > number_of_iterations) break;

		// integrate one iteration
		//std::cout << "Iteration " << iter << std::endl;
		for (uint step = 0; step < steps_per_iteration; step++)
		{
			// update odd values
			for (uint gpuIdx = 0; gpuIdx < gpuCount; ++gpuIdx)
			{
				hipSetDevice(gpuIdx);
				if (gpuIdx < gpuCount - 1)
					hipStreamWaitEvent(streamAndEvents[gpuIdx].kernelStream, streamAndEvents[gpuIdx + 1].backwardsEvent, 0);
				if (gpuIdx > 0)
					hipStreamWaitEvent(streamAndEvents[gpuIdx].kernelStream, streamAndEvents[gpuIdx - 1].forwardsEvent, 0);

				// Launch the CUDA kernel, even -> odd
				update << <dimGrids[gpuIdx], dimBlock, 0, streamAndEvents[gpuIdx].kernelStream >> > (d_oddPsis[gpuIdx], d_evenPsis[gpuIdx], d_pots[gpuIdx], d_blockDirs[gpuIdx], d_valueInds[gpuIdx], d_hodges[gpuIdx], g, dimensions[gpuIdx]);

				hipEventRecord(streamAndEvents[gpuIdx].kernelEvent, streamAndEvents[gpuIdx].kernelStream);
			}

			for (uint gpuIdx = 0; gpuIdx < gpuCount; ++gpuIdx)
			{
				hipSetDevice(gpuIdx);
				if (gpuIdx < gpuCount - 1)
				{
					hipStreamWaitEvent(streamAndEvents[gpuIdx].forwardsStream, streamAndEvents[gpuIdx].kernelEvent, 0);
					hipMemcpy3DAsync(&oddMemcpiesFrom[gpuIdx], streamAndEvents[gpuIdx].forwardsStream);
					hipEventRecord(streamAndEvents[gpuIdx].forwardsEvent, streamAndEvents[gpuIdx].forwardsStream);
				}
				if (gpuIdx > 0)
				{
					hipStreamWaitEvent(streamAndEvents[gpuIdx].backwardsStream, streamAndEvents[gpuIdx].kernelEvent, 0);
					hipMemcpy3DAsync(&oddMemcpiesTo[gpuIdx - 1], streamAndEvents[gpuIdx].backwardsStream);
					hipEventRecord(streamAndEvents[gpuIdx].backwardsEvent, streamAndEvents[gpuIdx].backwardsStream);
				}
			}

			// update even values
			for (uint gpuIdx = 0; gpuIdx < gpuCount; ++gpuIdx)
			{
				hipSetDevice(gpuIdx);
				if (gpuIdx < gpuCount - 1)
					hipStreamWaitEvent(streamAndEvents[gpuIdx].kernelStream, streamAndEvents[gpuIdx + 1].backwardsEvent, 0);
				if (gpuIdx > 0)
					hipStreamWaitEvent(streamAndEvents[gpuIdx].kernelStream, streamAndEvents[gpuIdx - 1].forwardsEvent, 0);

				// Launch the CUDA kernel, odd -> even
				update << <dimGrids[gpuIdx], dimBlock, 0, streamAndEvents[gpuIdx].kernelStream >> > (d_evenPsis[gpuIdx], d_oddPsis[gpuIdx], d_pots[gpuIdx], d_blockDirs[gpuIdx], d_valueInds[gpuIdx], d_hodges[gpuIdx], g, dimensions[gpuIdx]);

				hipEventRecord(streamAndEvents[gpuIdx].kernelEvent, streamAndEvents[gpuIdx].kernelStream);
			}

			for (uint gpuIdx = 0; gpuIdx < gpuCount; ++gpuIdx)
			{
				hipSetDevice(gpuIdx);
				if (gpuIdx < gpuCount - 1)
				{
					hipStreamWaitEvent(streamAndEvents[gpuIdx].forwardsStream, streamAndEvents[gpuIdx].kernelEvent, 0);
					hipMemcpy3DAsync(&evenMemcpiesFrom[gpuIdx], streamAndEvents[gpuIdx].forwardsStream);
					hipEventRecord(streamAndEvents[gpuIdx].forwardsEvent, streamAndEvents[gpuIdx].forwardsStream);
				}
				if (gpuIdx > 0)
				{
					hipStreamWaitEvent(streamAndEvents[gpuIdx].backwardsStream, streamAndEvents[gpuIdx].kernelEvent, 0);
					hipMemcpy3DAsync(&evenMemcpiesTo[gpuIdx - 1], streamAndEvents[gpuIdx].backwardsStream);
					hipEventRecord(streamAndEvents[gpuIdx].backwardsEvent, streamAndEvents[gpuIdx].backwardsStream);
				}
			}
		}
#if SAVE_PICTURE || SAVE_VOLUME
		// Copy back from device memory to host memory
		for (uint gpuIdx = 0; gpuIdx < gpuCount; ++gpuIdx)
		{
			hipSetDevice(gpuIdx);
			checkCudaErrors(hipMemcpy3DAsync(&evenPsiBackParams[gpuIdx]));
		}
#endif
	}
	for (uint gpuIdx = 0; gpuIdx < gpuCount; ++gpuIdx)
	{
		hipSetDevice(gpuIdx);
		hipDeviceSynchronize();
	}
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - startTime).count() * 1e-6;
	std::cout << "Duration for of the integration of one time unit: " << duration / number_of_iterations << std::endl;
	std::cout << "Total duration: " << duration << std::endl;

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch kernels (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	for (uint gpuIdx = 0; gpuIdx < gpuCount; ++gpuIdx)
	{
		checkCudaErrors(hipFree(d_cudaEvenPsis[gpuIdx].ptr));
		checkCudaErrors(hipFree(d_cudaOddPsis[gpuIdx].ptr));
		checkCudaErrors(hipFree(d_cudaPots[gpuIdx].ptr));
		checkCudaErrors(hipFree(d_blockDirs[gpuIdx]));
		checkCudaErrors(hipFree(d_valueInds[gpuIdx]));
		checkCudaErrors(hipFree(d_hodges[gpuIdx]));
	}
	return duration;
}

int main(int argc, char** argv)
{
#if LOAD_STATE_FROM_DISK
	VortexState state;
	state.load("state.dat");
	const ddouble eps = 1e-5 * state.searchFunctionMax();
	const ddouble maxr = state.searchMaxR(eps);
	const ddouble maxz = state.searchMaxZ(eps);
#else
	// preliminary vortex state to find vortex size
	VortexState state0;
	state0.setKappa(KAPPA);
	state0.setG(G);
	if (IS_3D) state0.setRange(0.0, 15.0, 35.0, 0.2, 0.2); // use this for 3d
	else state0.setRange(0.0, 15.0, 1.0, 0.2, 1.0); // use this for 2d
	state0.iterateSolution(potentialRZ, 10000, 1e-29);
	const ddouble eps = 1e-5 * state0.searchFunctionMax();
	const ddouble minr = state0.searchMinR(eps);
	ddouble maxr = state0.searchMaxR(eps);
	ddouble maxz = state0.searchMaxZ(eps);
	//std::cout << "maxf=" << 1e6*eps << " minr=" << minr << " maxr=" << maxr << " maxz=" << maxz << std::endl;

	// more accurate vortex state
	VortexState state;
	state.setKappa(KAPPA);
	state.setG(G);
	if (IS_3D) state.setRange(minr, maxr, maxz, 0.03, 0.03); // use this for 3d
	else state.setRange(minr, maxr, 1.0, 0.03, 1.0); // use this for 2d
	state.initialize(state0);
	state.iterateSolution(potentialRZ, 10000, 1e-29);
	state.save("state.dat");
	maxr = state.searchMaxR(eps);
	maxz = state.searchMaxZ(eps);
	//std::cout << "maxf=" << state.searchFunctionMax() << std::endl;
#endif
	uint gpuCount = (argc > 1) ? std::stoi(argv[1]) : 4;

	const int number_of_iterations = 1;
	const ddouble iteration_period = 1.0;
	ddouble blockScale = (argc > 2) ? std::stod(argv[2]) : 0.0391; // PIx2 / (20.0 * sqrt(state.integrateCurvature()));

	std::cout << gpuCount << " GPUs" << std::endl;
	std::cout << "kappa = " << KAPPA << ", " << state.getKappa() << std::endl;
	std::cout << "g = " << G << ", " << state.getG() << std::endl;
	std::cout << "iteration_period = " << iteration_period << std::endl;
	std::cout << "maxr = " << maxr << std::endl;
	std::cout << "maxz = " << maxz << std::endl;

	const ddouble EPSILON = 0.02;
	const ddouble TARGET = 7.1725;
	bool firstIter = true;
	while (true)
	{
		std::cout << "Block scale: " << blockScale << std::endl;

		auto time = integrateInTime(state, blockScale, Vector3(-maxr, -maxr, -maxz), Vector3(maxr, maxr, maxz), iteration_period, number_of_iterations, gpuCount, firstIter);

		auto timeDelta = time - TARGET;
		if (abs(timeDelta) < EPSILON)
		{
			break;
		}
		blockScale += std::pow(0.4 * timeDelta, 3);

		if (blockScale < 0)
		{
			std::cout << "SCALE BECAME NEGATIVE!" << std::endl;
			return 0;
		}
		std::cout << std::endl;
		firstIter = false;
	}

	return 0;
}
